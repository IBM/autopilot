#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <nvml.h>

#define MAX_BLOCKS 512
#define THREADS_PER_BLOCK 256
#define btoa(x) ((x)?"true":"false")

double cuda_dgemm(const char *, const char *, int *, int *, int *, double *, double *, int *, double *, int *, double *, double *, int *);
void cuda_dgemm_free();

#define CUDA_RC(rc) if( (rc) != hipSuccess ) \
  {printf("Error %s at %s line %d\n", hipGetErrorString(hipGetLastError()), __FILE__,__LINE__); exit(1);}

#define CUDA_CHECK()  if( (hipPeekAtLastError()) != hipSuccess )        \
  {printf("Error %s at %s line %d\n", hipGetErrorString(hipGetLastError()), __FILE__,__LINE__-1); exit(1);}

double walltime(void);

__global__ void daxpy(const double alpha, const double * x, double * y, int npts) 
{
   for (int i = blockDim.x * blockIdx.x + threadIdx.x;  i < npts; i += blockDim.x * gridDim.x) y[i] = alpha*x[i] + y[i];
}

static nvmlDevice_t nvmldevice;
static unsigned int temperature, power, smMHz;

int main(int argc, char * argv[])
{
   int i, d, npts, iter, maxiter, mydevice, numDevices;
   double * __restrict__ x, * __restrict__ y;
   double * dev_x, * dev_y;
   double * Amat, * Bmat, * Cmat;
   int m, n, k, lda, ldb, ldc;
   double alpha, beta;
   double  BW_pinned_h2d, BW_pageable_h2d, BW_pinned_d2h, BW_pageable_d2h, BW_daxpy, TFlops;
   double time1, time2;
   hipDeviceProp_t prop;
   double * metrics;
   nvmlDevice_t *device;
   unsigned int device_count;

   npts = 1024*1024*(1024/8);


   // initialize nvml
   if (NVML_SUCCESS != nvmlInit()) {
      fprintf(stderr, "failed to initialize NVML ... exiting\n");
   }   

   if (NVML_SUCCESS != nvmlDeviceGetCount(&device_count)) {
      fprintf(stderr, "nvmlDeviceGetCount failed ... exiting\n");
   }   

   device = (nvmlDevice_t *) malloc(device_count*sizeof(nvmlDevice_t));

   for (i = 0; i < device_count; i++) {
      if (NVML_SUCCESS != nvmlDeviceGetHandleByIndex(i, &device[i])) {
         fprintf(stderr, "nvmlDeviceGetHandleByIndex failed ... exiting\n");
      }   
   }

   // set matrix dimensions large enough to reach close to peak Flops
   m = 8192; n = 8192; k = 8192;
   Amat = (double *) malloc(m*k*sizeof(double));
   Bmat = (double *) malloc(k*n*sizeof(double));
   Cmat = (double *) malloc(m*n*sizeof(double));

#pragma omp parallel for
   for (i=0; i<(m*k); i++) Amat[i] = 1.2e-2*((double) (i%100));
#pragma omp parallel for
   for (i=0; i<(k*n); i++) Bmat[i] = 1.5e-3*((double) ((i + 100)%1000));
#pragma omp parallel for
   for (i=0; i<(m*n); i++) Cmat[i] = 1.5e-3*((double) ((i + 500)%1000));

   CUDA_RC(hipGetDeviceCount(&numDevices));


   metrics = (double *) malloc(numDevices*9*sizeof(double));
   y = (double *) malloc(npts*sizeof(double));
   
   bool* faulty = (bool*) malloc(numDevices*sizeof(bool));
   for (i = 0; i < numDevices; ++i)
      faulty[i] = false;


   for (d = 0; d < numDevices; d++) {
      mydevice = d; /*local_rank % numDevices;*/

      // assign nvmldevice to this rank's GPU
      nvmldevice = device[mydevice];

         CUDA_RC(hipSetDevice(mydevice));
         CUDA_RC(hipGetDeviceProperties(&prop, mydevice));
      
         // use pinned memory for x, pageable memory for y
         CUDA_RC(hipHostMalloc((void **)&x, npts*sizeof(double), hipHostMallocDefault));
         //   y = (double *) malloc(npts*sizeof(double));

         CUDA_RC(hipMalloc((void **)&dev_x, npts*sizeof(double)));
         CUDA_RC(hipMalloc((void **)&dev_y, npts*sizeof(double)));

         #pragma omp parallel for
         for (i=0; i<npts; i++) x[i] = (double) (i%10);
         #pragma omp parallel for
         for (i=0; i<npts; i++) y[i] = (double) (i%100);

         alpha = 3.0;
         maxiter = 20;

         

         time1 = walltime();
         CUDA_RC(hipMemcpy(dev_x, x, npts*sizeof(double), hipMemcpyHostToDevice));
         CUDA_RC(hipDeviceSynchronize());
         time2 = walltime();

         BW_pinned_h2d = 8.0e-9*((double) npts)/(time2 - time1);
         metrics[9*d+0] = BW_pinned_h2d;
         // Check here for low values in pinned h2d
         if (BW_pinned_h2d < 4)
            faulty[d] = true;

         time1 = walltime();
         CUDA_RC(hipMemcpy(dev_y, y, npts*sizeof(double), hipMemcpyHostToDevice));
         CUDA_RC(hipDeviceSynchronize());
         time2 = walltime();

         BW_pageable_h2d = 8.0e-9*((double) npts)/(time2 - time1);
         metrics[9*d+1] = BW_pageable_h2d;
   
         time1 = walltime();
         CUDA_RC(hipMemcpy(x, dev_x, npts*sizeof(double), hipMemcpyDeviceToHost));
         CUDA_RC(hipDeviceSynchronize());
         time2 = walltime();

         BW_pinned_d2h = 8.0e-9*((double) npts)/(time2 - time1);
         metrics[9*d+2] = BW_pinned_d2h;
         

         time1 = walltime();
         CUDA_RC(hipMemcpy(y, dev_y, npts*sizeof(double), hipMemcpyDeviceToHost));
         CUDA_RC(hipDeviceSynchronize());
         time2 = walltime();

         BW_pageable_d2h = 8.0e-9*((double) npts)/(time2 - time1);
         metrics[9*d+3] = BW_pageable_d2h;

         int threadsPerBlock = THREADS_PER_BLOCK;
         int numBlocks = (npts + threadsPerBlock - 1) / threadsPerBlock;
         if (numBlocks > MAX_BLOCKS) numBlocks = MAX_BLOCKS;

         time1 = walltime();
         for (iter=0; iter<maxiter; iter++) {
            daxpy<<<numBlocks, threadsPerBlock>>>(alpha, dev_x, dev_y, npts);
            CUDA_CHECK();
         }
         CUDA_RC(hipDeviceSynchronize());
         time2 = walltime();

         BW_daxpy = 3.0*8.0e-9*((double) npts)*((double) maxiter)/(time2 - time1);
         metrics[9*d+4] = BW_daxpy;
         if(BW_daxpy < 1300)
            faulty[d] = true;

         //   free(y);
         CUDA_RC(hipHostFree(x));
         CUDA_RC(hipFree(dev_x));
         CUDA_RC(hipFree(dev_y));

         beta = 0.0; lda = m; ldb = k; ldc = m;
         TFlops = cuda_dgemm("N", "N", &m, &n, &k, &alpha, Amat, &lda, Bmat, &ldb, &beta, Cmat, &ldc);
         cuda_dgemm_free();
         metrics[9*d+5] = TFlops;
         if(TFlops < 16)
            faulty[d] = true;

         metrics[9*d+6] = (double) temperature;
         metrics[9*d+7] = 1.0e-3*((double) power);  // convert to Watts
         metrics[9*d+8] = (double) smMHz;
   }
   printf(" GPU H2D(p)  H2D   D2H(p)  D2H   daxpy  dgemm   temp     power     smMHz\n");
   for (d = 0; d < numDevices; d++) {    
      printf("%3d %6.2lf %6.2lf %6.2lf %6.2lf %7.2lf %6.2lf %6.0lf %8.0lf %8.0lf\n", 
               d, metrics[9*d], metrics[9*d+1], metrics[9*d+2], metrics[9*d+3], metrics[9*d+4], metrics[9*d+5], metrics[9*d+6], metrics[9*d+7], metrics[9*d+8]);
   }
   printf("Summary of GPU errors:");
   bool allgood = true;
   for (d = 0; d < numDevices; d++) {
      if (faulty[d]) {
         allgood = false;
         printf("GPU %d -- H2D(p): %f; daxpy: %f; dgemm: %f", d, metrics[9*d+0], metrics[9*d+4], metrics[9*d+5]);
      }
   }
   if (allgood) {
      printf(" NONE ");
   }
   free(y);
   free(metrics);
   free(faulty);
   return 0;
}

double walltime(void)
{
  double elapsed;
  struct timeval tv;
  gettimeofday(&tv,NULL);
  elapsed = ((double) tv.tv_sec) + 1.0e-6*((double) tv.tv_usec);
  return elapsed;
}


// variables for cublas dgemm wrapper
static double * d_A, * d_B, * d_C;
static hipblasHandle_t handle;

// use the Fortran dgemm argument list
double cuda_dgemm(const char * transa, const char * transb, int * m, int * n, int * k, 
                  double * alpha, double * A, int * lda, double * B, int * ldb, 
                  double * beta, double * C, int * ldc)
{
   int M, N, K, LDA, LDB, LDC;
   int asize, bsize, csize;
   double time1, time2, TFlops;
   hipblasOperation_t opA, opB;
   int iter, maxiter = 400, sample_iter = 350;

   M = *m; N = *n; K = *k;
   LDA = *lda; LDB = *ldb; LDC = *ldc;

   asize = M*K;
   bsize = K*N;
   csize = M*N;

   hipblasCreate(&handle);
   hipMalloc((void **)&d_A, asize*sizeof(double));
   hipMalloc((void **)&d_B, bsize*sizeof(double));
   hipMalloc((void **)&d_C, csize*sizeof(double));

   hipblasSetVector(asize, sizeof(double), A, 1, d_A, 1);
   hipblasSetVector(bsize, sizeof(double), B, 1, d_B, 1);
   hipblasSetVector(csize, sizeof(double), C, 1, d_C, 1);

   if      (transa[0] == 'n' || transa[0] == 'N') opA = HIPBLAS_OP_N;
   else if (transa[0] == 't' || transa[0] == 'T') opA = HIPBLAS_OP_T;

   if      (transb[0] == 'n' || transb[0] == 'N') opB = HIPBLAS_OP_N;
   else if (transb[0] == 't' || transb[0] == 'T') opB = HIPBLAS_OP_T;


   // call one time outside the timers, then time it
   hipblasDgemm(handle, opA, opB, M, N, K, alpha, d_A, LDA, d_B, LDB, beta, d_C, LDC);
   hipDeviceSynchronize();

   time1 = walltime();
   for (iter = 0; iter < maxiter; iter++) {
      hipblasDgemm(handle, opA, opB, M, N, K, alpha, d_A, LDA, d_B, LDB, beta, d_C, LDC);
      if (iter == sample_iter) {
         if (NVML_SUCCESS != nvmlDeviceGetTemperature(nvmldevice, NVML_TEMPERATURE_GPU, &temperature)) temperature = 0; 
         if (NVML_SUCCESS != nvmlDeviceGetPowerUsage(nvmldevice, &power)) power = 0;
         if (NVML_SUCCESS != nvmlDeviceGetClockInfo(nvmldevice, NVML_CLOCK_SM, &smMHz)) smMHz = 0;
      }
      hipDeviceSynchronize();
   }
   time2 = walltime();
   TFlops = 2.0e-12*((double) maxiter)*((double) M)*((double) N)*((double) K)/(time2 - time1);

   hipMemcpy(C, d_C, csize*sizeof(double), hipMemcpyDeviceToHost);

   return TFlops;
}

void cuda_dgemm_free()
{
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
   hipblasDestroy(handle);
   return;
}
